#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 2048

using namespace std;


__global__ void mul_matrix(double* matrix_1, double* matrix_2, double* matrix_mul) {
    double value;
    int start_row = blockIdx.x, count_blocks = gridDim.x;
    int start_col = threadIdx.x, cout_thread = blockDim.x;

    for (int i = start_row; i < N; i += count_blocks)
        for (int j = start_col; j < N; j += cout_thread) {
            value = 0;
            for (int k = 0; k < N; ++k)
                value += matrix_1[i * N + k] * matrix_2[k * N + j];
            matrix_mul[i * N + j] = value;
        }
}

int main() {
    double* matrix_1, * matrix_2, * matrix_mul;
    int size = N * N * sizeof(double);
    hipEvent_t start, stop;
    float gpu_time;

    hipMallocManaged(&matrix_1, size);
    hipMallocManaged(&matrix_2, size);
    hipMallocManaged(&matrix_mul, size);

    for (int i = 0; i < N * N; ++i)
        matrix_1[i] = matrix_2[i] = 2;

    int _blocks = 32, _threads = 1024;
    dim3 threads(_threads);
    dim3 blocks(_blocks);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    mul_matrix << <blocks, threads >> > (matrix_1, matrix_2, matrix_mul);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Cout blocks = %i , count threads = %i , time = %f", _blocks, _threads, gpu_time);

    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(matrix_1); hipFree(matrix_2); hipFree(matrix_mul);
}